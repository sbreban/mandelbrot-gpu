#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "util.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <sys/timeb.h>

using namespace std;

#define cudaAssertSuccess(ans) { _cudaAssertSuccess((ans), __FILE__, __LINE__); }

__device__ __constant__ unsigned char d_const_colormap[(MAX_ITERATION + 1) * 3 * sizeof(unsigned char)];

__global__ void generate_image(unsigned char *d_image, unsigned char *d_colormap);

inline void _cudaAssertSuccess(hipError_t code, char *file, int line) {
	if(code != hipSuccess) {
		fprintf(stderr, "_cudaAssertSuccess: %s %s %d\n", hipGetErrorString(code), file, line);
		exit(code);
	}
}

__global__ void generate_image(unsigned char *d_image, unsigned char *d_colormap) {
	double c_re, c_im, x, y, x_new;
	int row, col, idx, iteration;

	int width = WIDTH;
	int height = HEIGHT;
	int max = MAX_ITERATION;

	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	idx = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

	if (idx >= width * height) return;

	for (int i = 0; i < PIXELS; i++) {
		int new_idx = PIXELS * idx + i;
		row = new_idx / WIDTH;
		col = new_idx % WIDTH;

		c_re = (col - width / 2.0)*4.0 / width;
		c_im = (row - height / 2.0)*4.0 / width;
		x = 0, y = 0;
		iteration = 0;
		while (x*x + y*y <= 4 && iteration < max) {
			x_new = x*x - y*y + c_re;
			y = 2 * x*y + c_im;
			x = x_new;
			iteration++;
		}
		if (iteration > max) {
			iteration = max;
		}
		d_image[4 * new_idx + 0] = d_colormap[iteration * 3 + 0];
		d_image[4 * new_idx + 1] = d_colormap[iteration * 3 + 1];
		d_image[4 * new_idx + 2] = d_colormap[iteration * 3 + 2];
		d_image[4 * new_idx + 3] = 255;
	}
}

__global__ void generate_image(unsigned char *d_image) {
	double c_re, c_im, x, y, x_new;
	int row, col, idx, iteration;

	int width = WIDTH;
	int height = HEIGHT;
	int max = MAX_ITERATION;

	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	idx = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

	if(idx >= width * height) return;

	for(int i = 0; i < PIXELS; i++) {
		int new_idx = PIXELS * idx + i;
		row = new_idx / WIDTH;
		col = new_idx % WIDTH;

		c_re = (col - width / 2.0)*4.0 / width;
		c_im = (row - height / 2.0)*4.0 / width;
		x = 0, y = 0;
		iteration = 0;
		while(x*x + y*y <= 4 && iteration < max) {
			x_new = x*x - y*y + c_re;
			y = 2 * x*y + c_im;
			x = x_new;
			iteration++;
		}
		if(iteration > max) {
			iteration = max;
		}
		d_image[4 * new_idx + 0] = d_const_colormap[iteration * 3 + 0];
		d_image[4 * new_idx + 1] = d_const_colormap[iteration * 3 + 1];
		d_image[4 * new_idx + 2] = d_const_colormap[iteration * 3 + 2];
		d_image[4 * new_idx + 3] = 255;
	}
}

void fractals(unsigned char *image, unsigned char *colormap, double *times) {
	unsigned char *d_image, *d_colormap;
	struct timeb start[REPEAT], end[REPEAT], before_data_send, after_data_send;
	char path[255];
	double data_send_time;
	dim3 grid(GRID_SIZE_X, GRID_SIZE_Y);
	dim3 block(BLOCK_SIZE_X, BLOCK_SIZE_Y);
	hipError_t cudaStatus;
	ftime(&before_data_send);

	cudaStatus = hipSetDevice(0);
	if(cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error_No_Free;
	}

	cudaStatus = hipMalloc(&d_image, WIDTH * HEIGHT * 4 * sizeof(unsigned char));
	if(cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error_No_Free;
	}

	if (USE_GLOBAL_MEMORY == 0) {
		cudaStatus = hipMalloc(&d_colormap, (MAX_ITERATION + 1) * 3 * sizeof(unsigned char));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error_Free_Image;
		}

		cudaStatus = hipMemcpy(d_colormap, colormap, (MAX_ITERATION + 1) * 3 * sizeof(unsigned char), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}
	} else {
		cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(d_const_colormap), colormap, (MAX_ITERATION + 1) * 3 * sizeof(unsigned char));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}
	}

	memset(image, 0, WIDTH * HEIGHT * 4 * sizeof(unsigned char));
	hipMemcpy(d_image, image, WIDTH * HEIGHT * 4 * sizeof(unsigned char), hipMemcpyHostToDevice);
	if(cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	ftime(&after_data_send);

	data_send_time = after_data_send.time - before_data_send.time + ((double)after_data_send.millitm - (double)before_data_send.millitm) / 1000.0;
	
	for(int i = 0; i < REPEAT; i++) {

		ftime(&start[i]);

		if (USE_GLOBAL_MEMORY == 0) {
			generate_image <<<grid, block >>> (d_image, d_colormap);
		}
		else {
			generate_image <<<grid, block >>> (d_image);
		}

		cudaStatus = hipGetLastError();
		if(cudaStatus != hipSuccess) {
			fprintf(stderr, "fractal launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}

		cudaStatus = hipDeviceSynchronize();
		if(cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			goto Error;
		}

		cudaStatus = hipMemcpy(image, d_image, WIDTH * HEIGHT * 4 * sizeof(unsigned char), hipMemcpyDeviceToHost);
		if(cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		ftime(&end[i]);
		times[i] = data_send_time + end[i].time - start[i].time + ((double)end[i].millitm - (double)start[i].millitm) / 1000.0;

		sprintf(path, IMAGE, "gpu", i);
		save_image(path, image, WIDTH, HEIGHT);
		progress("gpu", i, times[i]);
	}
	Error:
		if (USE_GLOBAL_MEMORY == 0) {
			hipFree(d_colormap);
		}
	Error_Free_Image:
	hipFree(d_image);
	Error_No_Free:
}


int main(int argc, char** argv) {
	struct arg a;
	double *times = (double*)malloc(sizeof(double)*REPEAT);

	unsigned char *colormap = (unsigned char*)malloc((MAX_ITERATION + 1) * 3);
	unsigned char *image = (unsigned char*)malloc(WIDTH * HEIGHT * 4);
	init_colormap(MAX_ITERATION, colormap);
	
	fractals(image, colormap, times);
	getchar();
	report("gpu", times);

	free(image);
	free(colormap);
	
	return 0;
}